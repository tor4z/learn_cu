#include <iostream>
#include "arr_add.hpp"


double sum_cpu(double *array1, double *array2, int N)
{
    double result = 0.0;
    for(int i = 0; i < N; ++i) {
        result += array1[i];
        result += array2[i];
    }
    return result;
}


double sum_gpu(double *array1, double *array2, int N)
{
    size_t size = N * sizeof(double);
    double result = 0.0;
    double *array3 = new double[N];
    double *d_array1;
    double *d_array2;
    double *d_array3;

    hipMalloc(&d_array1, size);
    hipMemcpy(d_array1, array1, size, hipMemcpyHostToDevice);

    hipMalloc(&d_array2, size);
    hipMemcpy(d_array2, array2, size, hipMemcpyHostToDevice);

    hipMalloc(&d_array3, size);
    hipMemcpy(d_array3, array3, size, hipMemcpyHostToDevice);

    sum_cuda<<<1, MAX_THREAD>>>(d_array1, d_array2, d_array3, N);

    hipMemcpy(array3, d_array3, size, hipMemcpyDeviceToHost);

    for(int i = 0; i< N; ++i)
        result += array3[i];

    free(array3);
    hipFree(d_array1);
    hipFree(d_array2);
    hipFree(d_array3);
    return result;
}


int main()
{
    int N = 100000001;
    double *array1 = new double[N];
    double *array2 = new double[N];

    // fill array
    for(int i = 0; i < N; ++i)
        array1[i] = (double)i;
        for(int i = 0; i < N; ++i)
        array2[i] = (double)i;

    std::cout << "Learn CUDA Programming" << std::endl;
    std::cout << "sum of array on cpu " << sum_cpu(array1, array2, N) << std::endl;
    std::cout << "sum of array on gpu " << sum_gpu(array1, array2, N) << std::endl;
}
