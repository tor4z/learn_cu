#include "hip/hip_runtime.h"
#include "arr_add.hpp"


__global__ void sum_cuda(double *array1,
                         double *array2,
                         double *array3,
                         int N)
{
    int i = threadIdx.x;
    for (int j = 0; j < (N / MAX_THREAD) && (i + j * MAX_THREAD) < N; ++j) {
        array3[i + j * MAX_THREAD] = 
            array1[i + j * MAX_THREAD] + array2[i + j * MAX_THREAD];
    }
}
